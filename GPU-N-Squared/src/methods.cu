#include "hip/hip_runtime.h"
#include <GL/gl.h>
#include <GL/freeglut.h>
#include <math.h>
#include <bits/stdc++.h>

#include <algorithm>

#include <string>
#include "headers.h"

#ifdef GPU
#if 0 
double *g_pos_x = nullptr;
double *g_pos_y = nullptr;
double *g_pos_next_x = nullptr;
double *g_pos_next_y = nullptr;
char *g_bm = nullptr;
#endif

__device__ __host__ void print_pos(double*, double*, int);


__device__ __host__ void g_move(unsigned int, double *, double *, double *, 
							double *, double * , double *, int *, double );
__device__ __host__ bool _g_move(unsigned int , double *, double *, double *, double *, double );

// This is the move that is launched from CPU and GPU runs it for each cell
__global__ void move_kernel(double *position_x, double *position_y, double *position_next_x, 
	double *position_next_y, double *velocity_x, double *velocity_y, int *status, int pop_size, double limit)
{
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= pop_size) return;
    if(status[index] == G_STOP){
	    status[index] = G_READY;
	    return;
	  }

	  if(status[index] == G_READY){
	    status[index] = G_RUNNING;
	  }

	  double tmp = position_x[index] + velocity_x[index];
	  if(tmp > limit || tmp < -limit){
	        velocity_x[index] = -velocity_x[index];
	  }
	  tmp = position_y[index] + velocity_y[index];
	  if(tmp > limit || tmp < -limit){
	        velocity_y[index] = -velocity_y[index];
	  }
	  position_x[index] += velocity_x[index];
	  position_y[index] += velocity_y[index];
	  
	  /* update pos_next after real movement */
	  position_next_x[index] = position_x[index];
	  position_next_y[index] = position_y[index];
     // printf("%d, %f %f\n", index, position_x[index], position_y[index]);
}

__global__ void move_prediction_kernel(double *position_next_x, double *position_next_y, double *velocity_x, double *velocity_y, int pop_size, double limit)
{
	unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index >= pop_size) return;

	double tmp = position_next_x[index] + velocity_x[index];
	if(tmp > limit || tmp < -limit){
		velocity_x[index] = -velocity_x[index];
	}
	tmp = position_next_y[index] + velocity_y[index];
	if(tmp > limit || tmp < -limit){
	    velocity_y[index] = -velocity_y[index];
	}
	position_next_x[index] += velocity_x[index];
	position_next_y[index] += velocity_y[index];
}

__device__ __host__
g_linked_tree::g_linked_tree(int id, int r, int n)
{
	this->id = id;
	root = r;
	previous = -1;
	node = n;
	branch = 1;
	branch_dist = 0;
}

__device__ __host__
void g_form_path(unsigned int linked1, unsigned int linked2, unsigned int finder, 
	struct g_linked_tree* g_trees, int* g_trees_next, int *status, double* velocity_x, double* velocity_y, unsigned int total_size)
{
	unsigned int tmp = linked1;
	while (g_trees[tmp].node != g_trees[tmp].root) {
		status[g_trees[tmp].node] = G_PATH;
		tmp = g_trees[tmp].previous;
	}
	tmp = linked2;
	while (g_trees[tmp].node != g_trees[tmp].root) {
		status[g_trees[tmp].node] = G_PATH;
		tmp = g_trees[tmp].previous;
	}
	status[finder] = G_PATH;
	velocity_x[finder] = 0;
	velocity_y[finder] = 0;
	g_trees[finder].root = g_trees[linked1].root; 
	g_trees[finder].previous = linked1;
	g_trees[finder].node = finder;
	
	if (g_trees[finder].previous == -1) {
		g_trees[finder].branch = 1;
		g_trees[finder].branch_dist = 0;
	} else {
		g_trees_next[g_trees[finder].previous * total_size + g_trees[g_trees[finder].previous].end++] = finder;
		g_trees[finder].branch_dist = g_trees[g_trees[finder].previous].branch_dist+1;
		/* This link is a branch if it meets global variable branch_len */
		// branch_len is 5 fixed at GPU version
		if (g_trees[finder].branch_dist >= 5) {
			g_trees[finder].branch = 1;
			g_trees[finder].branch_dist = 0;
		} else {
			g_trees[finder].branch = 0;
		}
	}
	
	//printf("FOUND A PATH\n");
}
#endif

linked_tree::linked_tree(objective *r, linked_tree *p, drawable *n) {
	root = r;
	previous = p;
	node = n;
	if (p == NULL) {
		this->branch = true;
		this->branch_dist = 0;
	} else {
		previous->next.push_back(this);
		this->branch_dist = previous->branch_dist+1;
		/* This link is a branch if it meets global variable branch_len */
		if (branch_dist >= branch_len) {
			branch = true;
			branch_dist = 0;
		} else {
			branch = false;
		}
	}
}

/** Parent class for things that need to be displayed on screen **/
drawable::drawable(unsigned int dimension, double radius, double limit, unsigned int id, population& p)
{
	this->id = id;
	this->dimension = dimension;
	this->limit = limit;
	this->pos = vector<double>(dimension, 0);
	this->radius = radius;

	/* initialize coordinates randomly */
	random_device dev;
	mt19937 rng(dev());
	uniform_real_distribution<double> dist(0, limit);
	uniform_int_distribution<mt19937::result_type> sign(0, 1);

	for(unsigned int i = 0; i < dimension; ++i) {
		double val = dist(rng);
		this->pos[i] = sign(rng) ? val : -val;
#ifdef GPU
		if(i == 0){
			p.position_x[id] = pos[i];
		} else if(i == 1){
			p.position_y[id] = pos[i];
		}
#endif
	}
}

/* draw itself in OpenGL by filling with tiny triangles */
void drawable::draw(double r, double g, double b)
{
	unsigned int count = 20;
	GLfloat twicePi = 2.0f * M_PI;

	glBegin(GL_TRIANGLE_FAN);

		/* center */
#ifdef GPU
		//log_file << this->id << "\t" << g_pos_x[this->id] << "\t" << g_pos_y[this->id] 
			//<< "\t" << r << "\t" << g << "\t" << b << "\t" << std::endl;
		/*
		glVertex2f(g_pos_x[this->id], g_pos_y[this->id]);
		for(unsigned int i = 0; i <= count; ++i) {
			glVertex2f(g_pos_x[this->id] + (radius * cos(i * twicePi / count)), g_pos_y[this->id] + (radius * sin(i * twicePi / count)));
		}
		*/
#else
		glVertex2f(pos[0], pos[1]);

		for(unsigned int i = 0; i <= count; ++i) {
			glVertex2f(pos[0] + (radius * cos(i * twicePi / count)), pos[1] + (radius * sin(i * twicePi / count)));
		}
#endif
	glEnd();
}

objective::objective(unsigned int dimension, double radius, double limit, unsigned int id, population& p) : drawable(dimension, radius, limit, id, p) {
	/* Set my id */
	this->id = id;	
	/* Establish root of linked tree */
	this->link = new linked_tree(this, NULL, this);

#ifdef GPU
	this->g_link = id;
#endif
};

/* 
 * dimension: fix to 2 at the moment
 * radius: fix to 20 at the moment
 * limit: playground dimension limit, only square allowed, assume that you want a 1000*1000 square then limit should be 1000
 */
individual::individual(unsigned int dimension, double radius, double limit, unsigned int mode, unsigned int id, population& p) : drawable(dimension, radius, limit, id, p)
{
	this->id = id;
	this->dimension = dimension;
	this->limit = limit;
	this->pos = vector<double>(dimension, 0);
	this->pos_next = vector<double>(dimension, 0);
	this->velocity = vector<double>(dimension, 0);
	this->radius = radius;
	this->status = READY;
	this->mode = mode;
	this->id = id;


	/* initialize coordinates and velocities randomly */
	random_device dev;
	mt19937 rng(dev());
	uniform_real_distribution<double> dist(0, limit);
	uniform_int_distribution<mt19937::result_type> sign(0, 1);

	double fixed_velocity = ((double)limit / 10000.0);

	/* only works when dimension is 2 */
	/* not in use right noew */
	if(mode == LEFTMOST_INIT) {
		double val = dist(rng);
		this->pos[1] = this->pos_next[1] = sign(rng) ? val : -val;
		this->velocity[0] = fixed_velocity;
		this->pos[0] = this->pos_next[0] = -limit;
		this->velocity[1] = 0;

		return;
	}

	for(unsigned int i = 0; i < dimension; ++i) {
		double val = dist(rng);
		this->pos[i] = this->pos_next[i] = sign(rng) ? val : -val;
		this->velocity[i] = sign(rng) ? fixed_velocity : -fixed_velocity;
#ifdef GPU	
		if(i == 0){
			p.position_x[id] = pos[i];
			p.position_next_x[id] = pos[i];
			p.velocity_x[id] = velocity[i];
		} else if(i == 1){
			p.position_y[id] = pos[i];
			p.position_next_y[id] = pos[i];
			p.velocity_y[id] = velocity[i];
		}
#endif
	}
}

/* pure move 
 * return value: true means entity collides on walls
 */
bool individual::_move(vector<double>& pos)
{
	bool res = false;
	for(unsigned int i = 0; i < this->dimension; ++i){
		/* detect collision with walls */
		double tmp = pos[i] + this->velocity[i];	
		if(tmp > limit || tmp < -limit){
				velocity[i] = -velocity[i];
				res = true;
		}
		pos[i] += velocity[i];
	}

	return res;
}

/* movement with respect to velocity */
void individual::move()
{
	if(this->status == STOP){
		this->status = READY;
		return;
	}

	if(this->status == READY){
		this->status = RUNNING;
	}

	_move(this->pos);
	
	/* update pos_next after real movement */
	for(unsigned int i = 0; i < this->dimension; ++i){
		this->pos_next[i] = this->pos[i];
	}

}

/* predict movement with respect to velocity */
void individual::move_prediction()
{
	bool terminate = _move(this->pos_next);
	
	/* on leftmost mode, if entity reaches targets, terminate permanently */
	if(mode == LEFTMOST_INIT && terminate){
		this->status = TERMINATE;
		this->velocity = vector<double>(this->dimension, 0);
	}
}

/*
 * test base on pos_next because we make sure that there is no collision at initial
 * another: a specific entity to test collision
 */
#ifdef GPU
/* test collision base on given two indices */
__device__ __host__
bool g_if_collision(double* pos_x, double* pos_y, unsigned int first, unsigned int second, double first_radius, double second_radius, double sense_dist)
{
	double distance = 0;
	double x1, x2, y1, y2;
	
	x1 = pos_x[first];
	x2 = pos_x[second];
	y1 = pos_y[first];
	y2 = pos_y[second];
	
	double tmp = x1 - x2;
	distance += (tmp * tmp);
	tmp = y1 - y2;
	distance += (tmp * tmp);

	distance = sqrt(distance);

	return distance < first_radius + sense_dist + second_radius ? true : false;
}

bool population::g_if_collision(unsigned int first, unsigned int second, bool first_use_pos_next, bool second_use_pos_next, double first_radius, double second_radius)
{
	double distance = 0;
	double x1, x2, y1, y2;

	x1 = first_use_pos_next ? this->position_next_x[first] : this->position_x[first];
	x2 = second_use_pos_next ? this->position_next_x[second] : this->position_x[second];
	y1 = first_use_pos_next ? this->position_next_y[first] : this->position_y[first];
	y2 = second_use_pos_next ? this->position_next_y[second] : this->position_y[second];
	
	double tmp = x1 - x2;
	distance += (tmp * tmp);
	tmp = y1 - y2;
	distance += (tmp * tmp);

	distance = sqrt(distance);

	return distance < first_radius + second_radius ? true : false;
}

#if 0
__device__ __host__
void inner_collision_kernel(unsigned int pop_size, double radius, unsigned int i, char* res, double* pos_x, double* pos_y, char* d_bm)
{
	unsigned int j = blockDim.x * blockIdx.x + threadIdx.x;
    if (j <= i || j >= pop_size) return;
	
	*res = g_if_collision(pos_x, pos_y, i, j, radius, radius) ? 1 : *res;
	
}
#endif

__global__
void collision_kernel(unsigned int pop_size, double radius, char* res, double* pos_x, double* pos_y, char* d_bm)
{
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= pop_size) return;

#if 0	
	dim3 blocksPerGrid(ceil((pop_size)/10.0), 1, 1);
	dim3 threadsPerBlock(10, 1, 1);

	inner_collision_kernel<<<blocksPerGrid,threadsPerBlock>>>(pop_size, radius, i, res, pos_x, pos_y, d_bm);
#endif
	
	for(unsigned int j = i+1; j < pop_size; ++j){
		if(g_if_collision(pos_x, pos_y, i, j, radius, radius, 0)){
			d_bm[i] = 1;
			d_bm[j] = 1;
			*res = 1;
		}
	}
	//printf("collision kernel: %f %f\n", pos_x[i], pos_y[i]);
}

__global__
void collision_diff_kernel(unsigned int pop_size, unsigned int obj_size, double radius1, double radius2, char* res, double* pos_x, double* pos_y, char* d_bm)
{
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= pop_size) return;
	
	for(unsigned int j = 0; j < obj_size; ++j){
		if(g_if_collision(pos_x, pos_y, i, pop_size + j, radius1, radius2, 0)){
			d_bm[i] = 1;
			d_bm[pop_size+j] = 1;
			*res = 1;
		}
	}
	//printf("collision diff kernel: %d %f %f\n", i, pos_x[i], pos_y[i]);
}

__global__
void sense_kernel(unsigned int pop_size, double radius, 
	double* pos_x, double* pos_y, int *status, double sense_dist)
{
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= pop_size || status[i] == G_LINK || status[i] == G_PATH || status[i] == G_ON_OBJ) return;
	
	for(unsigned int j = i+1; j < pop_size; ++j){
		if(g_if_collision(pos_x, pos_y, i, j, radius, radius, sense_dist) && status[j] == G_LINK){
			status[i] = G_SENSE;
			//printf("Entity %d sensed entity %d\n", i, j);
		}
	}
	
}

__global__
void sense_diff_kernel(unsigned int pop_size, unsigned int obj_size, double radius1, double radius2, 
	double* pos_x, double* pos_y, int *status, double sense_dist)
{
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= pop_size || status[i] == G_LINK || status[i] == G_PATH) return;
	
	for(unsigned int j = 0; j < obj_size; ++j){
		if(g_if_collision(pos_x, pos_y, i, pop_size + j, radius1, radius2, sense_dist)){
			status[i] = G_ON_OBJ;
			//printf("Entity %d sensed objective %d\n", i, j);
		}
	}
	
}

__global__
void decide_kernel(unsigned int pop_size, double radius, struct g_linked_tree* g_trees, int* g_trees_next, 
	double* pos_x, double* pos_y, double* velocity_x, double* velocity_y, int *status, double sense_dist, unsigned int total_size)
{
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= pop_size || status[i] != G_SENSE) return;
	
	int another = -1;
	int k_actual = -1;
	for(unsigned int j = i+1; j < pop_size; ++j){
		if(g_if_collision(pos_x, pos_y, i, j, radius, radius, sense_dist) && status[j] == G_LINK){
			if(another != -1 && g_trees[another].root != g_trees[j].root){
				g_form_path(another, j, i, g_trees, g_trees_next, status, velocity_x, velocity_y, total_size);
				break;
			}
			if(g_trees[j].end < 1 || g_trees[j].branch){
				another = j;
				k_actual = j;
			}
		}
	}
	
	//printf("Entity %d is trying to link with entity %d\n", i, k_actual);
	
	if(status[i] == G_PATH){
		return;
	}
	
	if(another == -1){
		status[i] = G_RUNNING;
		return;
	}
	
	status[i] = G_LINK;
	velocity_x[i] = 0;
	velocity_y[i] = 0;
	g_trees[i].id = i;
	g_trees[i].root = g_trees[another].root;
	g_trees[i].previous = another;
	g_trees[i].node = i;
	
	if (g_trees[i].previous == -1) {
		g_trees[i].branch = 1;
		g_trees[i].branch_dist = 0;
	} else {
		g_trees_next[g_trees[i].previous * total_size + g_trees[g_trees[i].previous].end++] = i;
		g_trees[i].branch_dist = g_trees[g_trees[i].previous].branch_dist+1;
		/* This link is a branch if it meets global variable branch_len */
		// branch_len is 5 fixed at GPU version
		if (g_trees[i].branch_dist >= 5) {
			g_trees[i].branch = 1;
			g_trees[i].branch_dist = 0;
		} else {
			g_trees[i].branch = 0;
		}
	}
	
	//printf("Entity %d is linking with entity %d\n", i, k_actual);
}

__global__
void decide_diff_kernel(unsigned int pop_size, unsigned int obj_size, double radius1, double radius2, struct g_linked_tree* g_trees, int* g_trees_next,
	double* pos_x, double* pos_y, double* velocity_x, double* velocity_y, int *status, double sense_dist, unsigned int total_size)
{
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= pop_size || status[i] != G_ON_OBJ) return;
	
	int obj = -1;
	for(unsigned int j = 0; j < obj_size; ++j){
		if(g_if_collision(pos_x, pos_y, i, pop_size + j, radius1, radius2, sense_dist)){
			if(obj == -1){
				obj = j;
			}
		}
	}
	
	//printf("Entity %d is trying to link with objective %d\n", i, obj);
	
	if(obj == -1){
		status[i] = G_RUNNING;
		return;
	}
	
	status[i] = G_LINK;
	velocity_x[i] = 0;
	velocity_y[i] = 0;
	g_trees[i].id = i;
	g_trees[i].root = g_trees[obj].root;
	g_trees[i].previous = obj;
	g_trees[i].node = i;
	
	if (g_trees[i].previous == -1) {
		g_trees[i].branch = 1;
		g_trees[i].branch_dist = 0;
	} else {
		g_trees_next[g_trees[i].previous * total_size + g_trees[g_trees[i].previous].end++] = i;
		g_trees[i].branch_dist = g_trees[g_trees[i].previous].branch_dist+1;
		/* This link is a branch if it meets global variable branch_len */
		// branch_len is 5 fixed at GPU version
		if (g_trees[i].branch_dist >= 5) {
			g_trees[i].branch = 1;
			g_trees[i].branch_dist = 0;
		} else {
			g_trees[i].branch = 0;
		}
	}
	
	//printf("Entity %d is linking with objective %d\n", i, obj);
}
#endif

bool objective::if_collision(objective *another)
{
	double distance = 0;

	for(unsigned int i = 0; i < this->dimension; ++i){
		double tmp = this->pos[i] - another->pos[i];
		distance += (tmp * tmp);
	}

	distance = sqrt(distance);

	return distance < this->radius + another->radius ? true : false;
}

bool individual::if_collision(individual another)
{
	double distance = 0;

	for(unsigned int i = 0; i < this->dimension; ++i){
		double tmp = this->pos_next[i] - another.pos_next[i];
		distance += (tmp * tmp);
	}

	distance = sqrt(distance);

	return distance < this->radius + another.radius ? true : false;
}

bool individual::if_collision(objective *another)
{
	double distance = 0;

	for(unsigned int i = 0; i < this->dimension; ++i){
		double tmp = this->pos_next[i] - another->pos[i];
		distance += (tmp * tmp);
	}

	distance = sqrt(distance);

	return distance < this->radius + another->radius ? true : false;
}

bool individual::if_sense(individual another, double sense_dist) {
	double distance = 0;

	for(unsigned int i = 0; i < this->dimension; ++i){
		double tmp = this->pos_next[i] - another.pos[i];
		distance += (tmp * tmp);
	}

	distance = sqrt(distance);

	return distance < this->radius + sense_dist + another.radius ? true : false;	
}

bool individual::if_sense(objective *another, double sense_dist) {
	double distance = 0;

	for(unsigned int i = 0; i < this->dimension; ++i){
		double tmp = this->pos_next[i] - another->pos[i];
		distance += (tmp * tmp);
	}

	distance = sqrt(distance);

	return distance < this->radius + sense_dist + another->radius ? true : false;	
}

void individual::grid_coordinates(unsigned int &x, unsigned int &y, double limit, double cell) {
	x = (unsigned int)((this->pos[0]+limit)/cell);
	y = (unsigned int)((this->pos[1]+limit)/cell);	
}

void population::sense(double sense_dist)
{
	sense_objectives(sense_dist);
	sense_entities(sense_dist);
}

/* Is each entity within sensing distance of any of the objectives? */
void population::sense_objectives(double sense_dist)
{
#ifdef GPU
	double* d_position_x = thrust::raw_pointer_cast(&position_x[0]);
	double* d_position_y = thrust::raw_pointer_cast(&position_y[0]);
	int* d_status = thrust::raw_pointer_cast(&g_status[0]);

	dim3 blocksPerGrid(ceil(pop_size/10.0), 1, 1);
	dim3 threadsPerBlock(10, 1, 1);

	sense_diff_kernel<<<blocksPerGrid,threadsPerBlock>>>(pop_size, num_objs, this->entities[0].radius, this->objectives[0]->radius, 
		d_position_x, d_position_y, d_status, sense_dist);
	hipDeviceSynchronize();

#else
	for (unsigned int i = 0; i < this->pop_size; ++i){
		if (this->entities[i].status == LINK || this->entities[i].status == PATH) continue;
		/* Is an entity on top of an objective? */
		for (unsigned int j = 0; j < this->num_objs; ++j) {
			if (this->entities[i].if_sense(this->objectives[j], sense_dist)) {
				//std::cout << "Entity " << i << " is on objective " << j << std::endl;
				this->entities[i].status = ON_OBJ;
			}
		}
	}
#endif
}

/* Is each entity (who has not already sensed an objective, or is not already linked or
within a path) within sensing distance of at least one other linked entity? */
void population::sense_entities(double sense_dist)
{	
#ifdef GPU
	double* d_position_x = thrust::raw_pointer_cast(&position_x[0]);
	double* d_position_y = thrust::raw_pointer_cast(&position_y[0]);
	int* d_status = thrust::raw_pointer_cast(&g_status[0]);

	dim3 blocksPerGrid(ceil(pop_size/10.0), 1, 1);
	dim3 threadsPerBlock(10, 1, 1);

	sense_kernel<<<blocksPerGrid,threadsPerBlock>>>(pop_size, this->entities[0].radius, d_position_x, d_position_y, d_status, sense_dist);
	hipDeviceSynchronize();
#else
	unsigned int left_x, right_x, up_y, down_y;
	double l = this->dim_limit;
	double c = this->cell_size;
	double s = sense_dist + this->entities[0].radius*2;

	/* TODO: make it supports GPU */

	// Loop through each entity
	for(unsigned int i = 0; i < this->pop_size; ++i){
		/* Continue to next entity if in tree */

		if (this->entities[i].status == ON_OBJ ||
			this->entities[i].status == LINK ||
			this->entities[i].status == PATH)
			continue;

		// Each entity will look at their own cell, as well as adjacent cells (including diagonal)
		left_x = max(0, (int)(((this->entities[i].pos[0]+l)-s)/c));
		right_x = min((int)this->grid_size-1, (int)(((this->entities[i].pos[0]+l)+s)/c));
		down_y = max(0, (int)(((this->entities[i].pos[1]+l)-s)/c));
		up_y = min((int)this->grid_size-1, (int)(((this->entities[i].pos[1]+l)+s)/c));
		// Compare current entity to all other entities in current/adjacent cells
		for (unsigned int x = left_x; x <= right_x; x++) {
			for (unsigned int y = down_y; y <= up_y; y++) {
				// Iterate through all entities in comparison cell
				for (unsigned int k = 0; k < this->grid[x][y].size(); k++) {
					// j = index of comparison entity (index in this->entities)
					unsigned int j = this->grid[x][y][k];
					// Check if "sensing self"
					if (i == j) continue;
					//cout << i << " | " << j << endl;
					if (this->entities[i].if_sense(this->entities[j], sense_dist) && 
						this->entities[j].status == LINK) {
						//std::cout << "Entity " << i << " sensed entity " << j << std::endl;
						this->entities[i].status = SENSE;
					}
				}
			}
		}
	}
#endif
}

void population::decide(double sense_dist)
{
	decide_link_objective(sense_dist);
	decide_link_entity(sense_dist);	
}

/* An entity will always link with objectives */
void population::decide_link_objective(double sense_dist)
{
#ifdef GPU
	double* d_position_x = thrust::raw_pointer_cast(&position_x[0]);
	double* d_position_y = thrust::raw_pointer_cast(&position_y[0]);
	double* d_velocity_x = thrust::raw_pointer_cast(&velocity_x[0]);
	double* d_velocity_y = thrust::raw_pointer_cast(&velocity_y[0]);
	int* d_status = thrust::raw_pointer_cast(&g_status[0]);
	struct g_linked_tree* d_linked_tree = thrust::raw_pointer_cast(&g_trees[0]);
	int* g_next = thrust::raw_pointer_cast(&g_trees_next[0]);

	dim3 blocksPerGrid(ceil(pop_size/10.0), 1, 1);
	dim3 threadsPerBlock(10, 1, 1);

	decide_diff_kernel<<<blocksPerGrid,threadsPerBlock>>>(pop_size, num_objs, this->entities[0].radius, this->objectives[0]->radius, 
		d_linked_tree, g_next, d_position_x, d_position_y, d_velocity_x, d_velocity_y, d_status, sense_dist, total_size);
	hipDeviceSynchronize();
#else
	for (unsigned int i = 0; i < this->pop_size; ++i){
		/* If the entity sensed that it was on an objective */
		if (this->entities[i].status == ON_OBJ) {
			/* Find objective that this entity is on top of */
			objective *obj_tmp = NULL;
			for (unsigned int j = 0; j < this->num_objs; ++j) {
				if (this->entities[i].if_sense(this->objectives[j], sense_dist)) {
					if (obj_tmp == NULL) {
						obj_tmp = this->objectives[j];
					} else {
						/* Single link between objectives would go here */
					}
				}
			}
			/* not sure if this can happen, but return to running if so */
			if (obj_tmp == NULL) {
				this->entities[i].status = RUNNING;
				continue;
			}
			/* Otherwise this entity is now in a linked_tree */
			//std::cout << "Entity " << i << " is linking with objective " << obj_tmp->id << std::endl;
			this->entities[i].status = LINK;
			this->entities[i].velocity = vector<double>(this->dim, 0);
			this->entities[i].link = new linked_tree(obj_tmp->link->root, obj_tmp->link, &(this->entities[i]));
		}
	}
#endif
}

/* Entities will try to link with other linked entities if possible (free link or branch)
 If an entity can link with two linked entities from different linked_trees, a path is detected */
void population::decide_link_entity(double sense_dist)
{
#ifdef GPU
	double* d_position_x = thrust::raw_pointer_cast(&position_x[0]);
	double* d_position_y = thrust::raw_pointer_cast(&position_y[0]);
	double* d_velocity_x = thrust::raw_pointer_cast(&velocity_x[0]);
	double* d_velocity_y = thrust::raw_pointer_cast(&velocity_y[0]);
	int* d_status = thrust::raw_pointer_cast(&g_status[0]);
	struct g_linked_tree* d_linked_tree = thrust::raw_pointer_cast(&g_trees[0]);
	int* g_next = thrust::raw_pointer_cast(&g_trees_next[0]);

	dim3 blocksPerGrid(ceil(pop_size/10.0), 1, 1);
	dim3 threadsPerBlock(10, 1, 1);

	decide_kernel<<<blocksPerGrid,threadsPerBlock>>>(pop_size, this->entities[0].radius,
		d_linked_tree, g_next, d_position_x, d_position_y, d_velocity_x, d_velocity_y, d_status, sense_dist, total_size);
	hipDeviceSynchronize();
#else
	for (unsigned int i = 0; i < this->pop_size; ++i){
		if (this->entities[i].status == SENSE) {
			/* Find the first linked entity that is also free or able to branch */
			//std::cout << "Entity " << i << " is deciding what to do after sensing another linked entity" << std::endl;
			individual *another_tmp = NULL;
			unsigned int k_actual = -1;
			for (unsigned int k = 0; k < this->pop_size; ++k) {
				if (i == k) continue;
				if (this->entities[i].if_sense(this->entities[k], sense_dist) &&
					this->entities[k].status == LINK) {
					if (another_tmp != NULL && another_tmp->link->root != this->entities[k].link->root) {
						form_path(another_tmp, &(this->entities[k]), &(this->entities[i]));
						break;
					}
					if (this->entities[k].link->next.size() < 1 || this->entities[k].link->branch) {
						another_tmp = &(this->entities[k]);
						k_actual = k;
					}
				}
			}
			if (this->entities[i].status == PATH) {
				continue;
			}
			/* This can happen if two entities sense the same linked entity (one will link with it before the other) */
			if (another_tmp == NULL) {
				this->entities[i].status = RUNNING;
				continue;
			}
			//std::cout << "Entity " << i << " is linking with entity " << k_actual << std::endl;
			/* Entity is within sensing distance of a linked entity, so create another link */
			//another_tmp->link->free = false;
			this->entities[i].status = LINK;
			this->entities[i].velocity = vector<double>(this->dim, 0);
			this->entities[i].link = new linked_tree(another_tmp->link->root, another_tmp->link, &(this->entities[i]));
		}
	}
#endif	
}

/* test if collision exists otherwise update collision bitmap 
	comparisons are only made to nearby entities */

bool population::collision()
{

	bool res = false;
	unsigned int left_x, right_x, up_y, down_y;
	double l = this->dim_limit;
	double c = this->cell_size;
	double r = this->entities[0].radius*2;

	/* TODO: make it supports GPU */

#ifdef GPU
	double* d_position_x =  thrust::raw_pointer_cast(&position_next_x[0]);
	double* d_position_y =  thrust::raw_pointer_cast(&position_next_y[0]);
	char* d_bm =  thrust::raw_pointer_cast(&g_bm[0]);
	char* d_res;
	hipError_t err = hipMallocManaged((void **) &d_res, 1);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	*d_res = 0;

	dim3 blocksPerGrid(ceil(pop_size/10.0), 1, 1);
	dim3 threadsPerBlock(10, 1, 1);

	collision_kernel<<<blocksPerGrid,threadsPerBlock>>>(this->pop_size, this->entities[0].radius, d_res, d_position_x, d_position_y, d_bm);
	hipDeviceSynchronize();
	res = *d_res ? true : res;
	hipFree(d_res);

#else
	// Loop through each entity
	for(unsigned int i = 0; i < this->pop_size; ++i){
		// Each entity will look at their own cell, as well as adjacent cells (including diagonal)
		left_x = max(0, (int)((this->entities[i].pos[0]+l-r)/c));
		right_x = min((int)this->grid_size-1, (int)((this->entities[i].pos[0]+l+r)/c));
		down_y = max(0, (int)((this->entities[i].pos[1]+l-r)/c));
		up_y = min((int)this->grid_size-1, (int)((this->entities[i].pos[1]+l+r)/c));
		// Compare current entity to all other entities in current/adjacent cells
		for (unsigned int x = left_x; x <= right_x; x++) {
			for (unsigned int y = down_y; y <= up_y; y++) {
				// Iterate through all entities in comparison cell
				for (unsigned int k = 0; k < this->grid[x][y].size(); k++) {
					// j = index of comparison entity (index in this->entities)
					unsigned int j = this->grid[x][y][k];
					// Check if "comparing to self"
					if (i == j) continue;
					// If colliding with another, then do collision behavior
					if (this->entities[i].if_collision(this->entities[j])) {
						this->bm[i].bit=1;
						this->bm[j].bit=1;
						res = true;
					}
				}
			}
		}
	}
#endif

	return res;
}

/* customized collsion test only used after initialization */
bool population::init_collision()
{
	bool res = false;
	
#ifdef GPU

	double* d_position_x =  thrust::raw_pointer_cast(&position_x[0]);
	double* d_position_y =  thrust::raw_pointer_cast(&position_y[0]);
	char* d_bm =  thrust::raw_pointer_cast(&g_bm[0]);
	char* d_res;
	hipError_t err = hipMallocManaged((void **) &d_res, 1);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	*d_res = 0;
	hipStream_t streams[2];
	hipStreamCreate(&streams[0]);
	hipStreamCreate(&streams[1]);

	dim3 blocksPerGrid(ceil(pop_size/10.0), 1, 1);
	dim3 threadsPerBlock(10, 1, 1);

	// between entities
	collision_kernel<<<blocksPerGrid,threadsPerBlock, 0, streams[0]>>>(this->pop_size, this->entities[0].radius, d_res, d_position_x, d_position_y, d_bm);

	// between entities and objects
	collision_diff_kernel<<<blocksPerGrid,threadsPerBlock, 0, streams[1]>>>(this->pop_size, this->num_objs, this->entities[0].radius, this->objectives[0]->radius, d_res, d_position_x, d_position_y, d_bm);
	
	hipDeviceSynchronize();
	
	res = *d_res ? true : res;
	hipFree(d_res);

#else
	// between entities
	for(unsigned int i = 0; i < this->pop_size; ++i){
		for(unsigned int j = i+1; j < this->pop_size; ++j){
			if(this->entities[i].if_collision(this->entities[j])){
				this->bm[i].bit = 1;
				this->bm[j].bit = 1;
				res = true;
			}
		}
	}

	// between entities and objects
	for(unsigned int i = 0; i < this->pop_size; ++i){
		for(unsigned int j = 0; j < this->num_objs; ++j){
			if(this->entities[i].if_collision(this->objectives[j])){
				this->bm[i].bit = 1;
				this->bm[this->pop_size+j].bit = 1;
				res = true;
			}
		}
	}
#endif

	// between objects
	for(unsigned int i = 0; i < this->num_objs; ++i){
		for(unsigned int j = i+1; j < this->num_objs; ++j){
			if(this->objectives[i]->if_collision(this->objectives[j])){
				this->bm[this->pop_size+i].bit = 1;
				this->bm[this->pop_size+j].bit = 1;
#ifdef GPU
				g_bm[this->pop_size+i] = 1;
				g_bm[this->pop_size+j] = 1;
#endif
				res = true;
			}
		}
	}

	return res;
	
}

__global__
void first_adjustment_kernel(unsigned int pop_size, double* pos_x, double* pos_y,
	double* pos_next_x, double* pos_next_y, double* velocity_x, double* velocity_y, char* d_bm, double limit)
{
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= pop_size || d_bm[i] == 0) return;
	
	d_bm[i] = 0;
	velocity_x[i] = -velocity_x[i];
	velocity_y[i] = -velocity_y[i];
	pos_next_x[i] = pos_x[i];
	pos_next_y[i] = pos_y[i];
	
	double tmp = pos_next_x[i] + velocity_x[i];
	if(tmp > limit || tmp < -limit){
		velocity_x[i] = -velocity_x[i];
	}
	tmp = pos_next_y[i] + velocity_y[i];
	if(tmp > limit || tmp < -limit){
	    velocity_y[i] = -velocity_y[i];
	}
	pos_next_x[i] += velocity_x[i];
	pos_next_y[i] += velocity_y[i];
}

__global__
void second_adjustment_kernel(unsigned int pop_size, double* pos_x, double* pos_y,
	double* pos_next_x, double* pos_next_y, int* d_status, char* d_bm)
{
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= pop_size || d_bm[i] == 0) return;
	
	d_bm[i] = 0;
	pos_next_x[i] = pos_x[i];
	pos_next_y[i] = pos_y[i];
	d_status[i] = G_STOP;
}

/* adjuest velocity of each entity with respect to collision detection */
void population::adjustment()
{
	unsigned int retries = 0;

	/* retries 2 times because we only have 2 directions to go at the moment */
	while(collision() && retries++ < 2){
#ifdef GPU
		double* d_position_x = thrust::raw_pointer_cast(&position_x[0]);
		double* d_position_y = thrust::raw_pointer_cast(&position_y[0]);
		double* d_position_next_x = thrust::raw_pointer_cast(&position_next_x[0]);
		double* d_position_next_y = thrust::raw_pointer_cast(&position_next_y[0]);
		double* d_velocity_x = thrust::raw_pointer_cast(&velocity_x[0]);
		double* d_velocity_y = thrust::raw_pointer_cast(&velocity_y[0]);

		char* d_bm =  thrust::raw_pointer_cast(&g_bm[0]);

		dim3 blocksPerGrid(ceil(pop_size/10.0), 1, 1);
		dim3 threadsPerBlock(10, 1, 1);

		first_adjustment_kernel<<<blocksPerGrid,threadsPerBlock>>>(this->pop_size, d_position_x, d_position_y, 
			d_position_next_x, d_position_next_y, d_velocity_x, d_velocity_y, d_bm, this->limit);
		hipDeviceSynchronize();

#else
		for(unsigned int i = 0; i < this->pop_size; ++i){
			if(this->bm[i].bit){
				this->bm[i].bit = 0;
				/* simply reverse the direction at the moment */
				/* also update pos_next */
				/* TODO: should be calculate by genetic algorithm */
				for(unsigned int j = 0; j < this->entities[i].dimension; ++j){
					this->entities[i].velocity[j] = -this->entities[i].velocity[j];
					this->entities[i].pos_next[j] = this->entities[i].pos[j];
				}

				this->entities[i].move_prediction();
				
				/* FIXME: copy host back to device, shouldn't do that for final product */
			}
		}
#endif
	}

	/* collision still exists, stop the entities detected collision */
	retries = 0;
	while(collision() && retries < 2){
#ifdef GPU
		double* d_position_x = thrust::raw_pointer_cast(&position_x[0]);
		double* d_position_y = thrust::raw_pointer_cast(&position_y[0]);
		double* d_position_next_x = thrust::raw_pointer_cast(&position_next_x[0]);
		double* d_position_next_y = thrust::raw_pointer_cast(&position_next_y[0]);
		int* d_g_status = thrust::raw_pointer_cast(&g_status[0]);

		char* d_bm =  thrust::raw_pointer_cast(&g_bm[0]);

		dim3 blocksPerGrid(ceil(pop_size/10.0), 1, 1);
		dim3 threadsPerBlock(10, 1, 1);

		second_adjustment_kernel<<<blocksPerGrid,threadsPerBlock>>>(this->pop_size, d_position_x, d_position_y, 
			d_position_next_x, d_position_next_y, d_g_status, d_bm);
		hipDeviceSynchronize();
#else
		for(unsigned int i = 0; i < this->pop_size; ++i){
			if(this->bm[i].bit){
				this->bm[i].bit = 0;
				this->entities[i].status = STOP;
				/* reset pos_next */
				for(unsigned int j = 0; j < this->entities[i].dimension; ++j){
					this->entities[i].pos_next[j] = this->entities[i].pos[j];
				}
			}
		}
#endif
	}
	
	if(retries == 2){
		std:: cout << "No room for that much(big) entities! "
			   << "Program exit at this point because collision between entities could not be solved "
			   << "Please revise your arguments by "
			   << "decresing the [population size], [radius of entity] "
			   << "or incresing the [playground dimension]" 
			   << endl;
		exit(1);
	}
}

/* check if all entities terminate */
bool population::terminate()
{
	bool res = true;

	for(auto e : entities){
		if(e.status != TERMINATE){
			res = false;
			break;
		}
	}

	return res;
}

void population::form_path(individual *linked1, individual *linked2, individual *finder) {

	//linked_tree *prev;

	linked_tree *tmp = linked1->link;
	while (tmp->node != tmp->root) {
		((individual *)tmp->node)->status = PATH;
		tmp = tmp->previous;
	}
	tmp = linked2->link;
	while (tmp->node != tmp->root) {
		((individual *)tmp->node)->status = PATH;
		tmp = tmp->previous;
	}
	finder->status = PATH;
	finder->velocity = vector<double>(this->dim, 0);
	finder->link = new linked_tree(linked1->link->root, linked1->link, finder);
	//std::cout << "FOUND A PATH" << std::endl;
}

void population::init_grid(double radius, double dimension_limit) {
	this->cell_size = 1;
	while (cell_size < (radius*2)) { cell_size *= 2; }
	this->grid_size = ((dimension_limit*2)/cell_size)+1;

	this->grid = new vector<unsigned int> *[grid_size];
	for (unsigned int i = 0; i < grid_size; i++) {
		this->grid[i] = new vector<unsigned int> [grid_size];
		for (unsigned int j = 0; j < grid_size; j++) {
			this->grid[i][j] = vector<unsigned int>();
		}
	}
	//std::cout << "Grid is " << grid_size << " by " << grid_size << " with cell size " << cell_size << std::endl;
}

void population::clear_grid() {
	for (unsigned int i = 0; i < grid_size; i++) {
		for (unsigned int j = 0; j < grid_size; j++) {
			this->grid[i][j].clear();
		}
	}
}

void population::assign_to_grid() {
	unsigned int grid_x,grid_y;
	for (unsigned int i = 0; i < this->pop_size; i++) {
		// grid_x, grid_y are being passed by reference
		this->entities[i].grid_coordinates(grid_x, grid_y, this->dim_limit, this->cell_size);	
		this->grid[grid_x][grid_y].push_back(this->entities[i].id);
	}
}


#ifdef GPU
void gpu_uni_malloc(void **buf, size_t size)
{
	hipError_t err = hipMallocManaged(buf, size);
	if(err != hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
}
#endif

__global__ void draw_kernel(double *position_x, double *position_y, int *status, int pop_size, int num_objs)
{
	unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index >= pop_size+num_objs) return;
	else if (index >= pop_size) {
		printf("%d\t%f\t%f\t1.\t0.\t0.\n",index, position_x[index], position_y[index]);
	}
	else {
		if (status[index] == G_LINK)
			printf("%d\t%f\t%f\t0.\t1.\t0.\n",index, position_x[index], position_y[index]);
		else if (status[index] == G_PATH)
			printf("%d\t%f\t%f\t.5\t0.\t.5\n",index, position_x[index], position_y[index]);
		else
			printf("%d\t%f\t%f\t0.\t0.\t1.\n",index, position_x[index], position_y[index]);
	}

}

void population::draw() {
	double* d_position_x = thrust::raw_pointer_cast(&position_x[0]);
	double* d_position_y = thrust::raw_pointer_cast(&position_y[0]);
	double* d_velocity_x = thrust::raw_pointer_cast(&velocity_x[0]);
	double* d_velocity_y = thrust::raw_pointer_cast(&velocity_y[0]);
	int* d_g_status = thrust::raw_pointer_cast(&g_status[0]);


	dim3 blocksPerGrid(ceil((pop_size+num_objs)/16.0), 1, 1);
	dim3 threadsPerBlock(16, 1, 1);


	draw_kernel<<<blocksPerGrid,threadsPerBlock>>>(d_position_x, d_position_y, d_g_status, pop_size, num_objs);
	hipDeviceSynchronize();
}

/*
 * size: fix to 10 at the moment
 * others: same to the arguments of individual(...)
 */
population::population(unsigned int size, unsigned int dimension, double radius, double limit, unsigned int num_objectives, double objective_radius, unsigned int mode)
{
	this->pop_size = size;
	this->num_objs = num_objectives;
	this->dim = dimension;
	this->dim_limit = limit;


#ifdef GPU
	this->total_size = size + num_objectives;
	this->position_x = thrust::device_vector<double>(total_size, 0);
	this->position_y = thrust::device_vector<double>(total_size, 0);
	this->position_next_x = thrust::device_vector<double>(size, 0);
	this->position_next_y = thrust::device_vector<double>(size, 0);
	this->velocity_x = thrust::device_vector<double>(size, 0);
	this->velocity_y = thrust::device_vector<double>(size, 0);

	this->g_status = thrust::device_vector<double>(size, 2);
	this->g_bm = thrust::device_vector<char>(total_size, 0);
	this->limit = limit;
	
	this->g_trees = thrust::device_vector<struct g_linked_tree>(total_size, g_linked_tree(-1, -1, -1));
	this->g_trees_next = thrust::device_vector<int>(total_size*total_size, -1);
	
#if 0
	gpu_uni_malloc((void **) &g_pos_x, (size+num_objectives) * sizeof(double));
	gpu_uni_malloc((void **) &g_pos_y, (size+num_objectives) * sizeof(double));
	gpu_uni_malloc((void **) &g_pos_next_x, size * sizeof(double));
	gpu_uni_malloc((void **) &g_pos_next_y, size * sizeof(double));
	gpu_uni_malloc((void **) &g_bm, (size+num_objectives) * sizeof(char));
#endif
	
#endif

	for(unsigned int i = 0; i < size; ++i){
		this->entities.push_back(individual(dimension, radius, limit, mode, i, *this));
		this->bm.push_back(one_bit());
#ifdef GPU
		g_bm[i] = 0;
#endif
	}

	for(unsigned int i = 0; i < num_objectives; ++i) {
		objective *tmp = new objective(dimension, objective_radius, limit, size+i, *this);
		this->objectives.push_back(tmp);
		this->bm.push_back(one_bit());
#ifdef GPU
		g_bm[size+i] = 0;
		g_trees.push_back(g_linked_tree(size+i, size+i, size+i));
#endif
	}

	unsigned int retries = 0;
	/* make sure the population is initialized with no collision, give a retry limitation to prevent forever loop */
	while(init_collision() && retries++ < 99){
		/* TODO: GPU version */
		for(unsigned int i = 0; i < size; ++i){
#ifdef GPU
			if(g_bm[i]){
				g_bm[i] = 0;
#else
			if(this->bm[i].bit){
				this->bm[i].bit = 0;
#endif
				this->entities[i] = individual(dimension, radius, limit, mode, i, *this);
			}
		}

		for(unsigned int i = 0; i < num_objectives; ++i){
#ifdef GPU
			if(g_bm[size+i]){
				g_bm[size+i] = 0;
#else
			if(this->bm[size+i].bit){
				this->bm[size+i].bit = 0;
#endif
				/* FIXME: memory leak here, we need a destructor to make it works properly */
				this->objectives[i] = new objective(dimension, objective_radius, limit, size+i, *this);
			}
		}
	}

	if(retries == 100){
		std:: cout << "No room for that much(big) entities! "
			   << "Please revise your arguments by "
			   << "decresing the [population size], [radius of entity] "
			   << "or incresing the [playground dimension]" 
			   << endl;
		exit(1);
	}

	this->init_grid(radius, limit);
	this->clear_grid();
	this->assign_to_grid();

}


#ifdef GPU
void population::birth_robot()
{
  position_x.push_back(2.0);
  position_y.push_back(2.5);

  // velocity_x.push_back(2.0f * ((double)rand() / (double)RAND_MAX) - 1.0f);
  // velocity_y.push_back(2.0f * ((double)rand() / (double)RAND_MAX) - 1.0f);

  velocity_x.push_back(1.0);
  velocity_y.push_back(1.0);

  g_status.push_back(2);
}

void population::advance_robot()
{
  // As we cannot send device vectors to the move (as device_vector is at
  // the end of the day a GPU structure abstraction in CPU) we have to get the
  // pointer in GPU memory in order for the move to know where to start 
  // reading the double arrays from.

  double* d_position_x = thrust::raw_pointer_cast(&position_x[0]);
  double* d_position_y = thrust::raw_pointer_cast(&position_y[0]);
  double* d_position_next_x = thrust::raw_pointer_cast(&position_next_x[0]);
  double* d_position_next_y = thrust::raw_pointer_cast(&position_next_y[0]);
  double* d_velocity_x = thrust::raw_pointer_cast(&velocity_x[0]);
  double* d_velocity_y = thrust::raw_pointer_cast(&velocity_y[0]);
  int* d_g_status = thrust::raw_pointer_cast(&g_status[0]);


  dim3 blocksPerGrid(ceil(pop_size/10.0), 1, 1);
  dim3 threadsPerBlock(10, 1, 1);


  move_kernel<<<blocksPerGrid,threadsPerBlock>>>(d_position_x, d_position_y, d_position_next_x,
   d_position_next_y, d_velocity_x, d_velocity_y, d_g_status, pop_size, this->limit);
  hipDeviceSynchronize();

}

void population::predict_robot()
{
  double* d_position_next_x = thrust::raw_pointer_cast(&position_next_x[0]);
  double* d_position_next_y = thrust::raw_pointer_cast(&position_next_y[0]);
  double* d_velocity_x = thrust::raw_pointer_cast(&velocity_x[0]);
  double* d_velocity_y = thrust::raw_pointer_cast(&velocity_y[0]);

  dim3 blocksPerGrid(ceil(pop_size/10.0), 1, 1);
  dim3 threadsPerBlock(10, 1, 1);

  move_prediction_kernel<<<blocksPerGrid,threadsPerBlock>>>(d_position_next_x, d_position_next_y, d_velocity_x, d_velocity_y, pop_size, this->limit);
  hipDeviceSynchronize();
}

__device__ __host__ void print_pos(double* position_x, double* position_y, int i){
  printf("%f %f\n", position_x[i], position_y[i]);
}

/* pure move 
 * return value: true means entity collides on walls
 */
__device__ __host__ 
bool _g_move(unsigned int index, double *position_x, double *position_y, 
  double * velocity_x, double *velocity_y, double limit)
{
  bool res = false;
  double tmp = position_x[index] + velocity_x[index];
  if(tmp > limit || tmp < -limit){
        velocity_x[index] = -velocity_x[index];
        res = true;
  }
  tmp = position_y[index] + velocity_y[index];
  if(tmp > limit || tmp < -limit){
        velocity_y[index] = -velocity_y[index];
        res = true;
  }
  position_x[index] += velocity_x[index];
  position_y[index] += velocity_y[index];
  return res;
}

/* movement with respect to veclocity */
__device__ __host__ 
void g_move(unsigned int index, double *position_x, double *position_y, double *position_next_x, 
	double *position_next_y, double * velocity_x, double *velocity_y, int *status, double limit)
{
  if(status[index] == 1){
    status[index] = 2;
    return;
  }

  if(status[index] == 2){
    status[index] = 3;
  }

  double tmp = position_x[index] + velocity_x[index];
  if(tmp > limit || tmp < -limit){
        velocity_x[index] = -velocity_x[index];
  }
  tmp = position_y[index] + velocity_y[index];
  if(tmp > limit || tmp < -limit){
        velocity_y[index] = -velocity_y[index];
  }
  position_x[index] += velocity_x[index];
  position_y[index] += velocity_y[index];
  
  /* update pos_next after real movement */
  position_next_x[index] = position_x[index];
  position_next_y[index] = position_y[index];
}
#endif
